#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <sstream>
#include <vector>
#include "Compressor.cuh"
#include "DataTypes.h"
#include "Helpers.h"
#include "TagReader.h"
#include "TagTable.h"
#include "Timer.h"
#include "VolumeDesc.h"

#ifdef USE_TIMER // timer is in "Compressor.cuh"
float timerTotal = 0;
#include <iostream>
#endif

using namespace std;

// container for block of voxels
vector<BlockPlane> blockPlane;

// lookup between string names and uchar IDs
TagTable globalTT = TagTable();

// read file first line to get description
// and copy to device
__host__ void readFileDescription()
{
    // Get the description string from setup TagReader
    string description = TagReader::setup();

    // Parse and save the compression dimensions
    stringstream ss(description);
    char ignore;
    ss >> ignore // discard first char which is '#'
        >> VolumeDesc::totalSize.x
        >> VolumeDesc::totalSize.y
        >> VolumeDesc::totalSize.z
        >> VolumeDesc::pBlockSize.x
        >> VolumeDesc::pBlockSize.y
        >> VolumeDesc::pBlockSize.z;

    // Calculate the total number of voxels in the volume
    VolumeDesc::totalNumVoxels = getVolume(VolumeDesc::totalSize);

    // Calculate the total number of blocks in a parent block
    VolumeDesc::pBlockNumVoxels = getVolume(VolumeDesc::pBlockSize);

    // how many parent blocks fit in each dimension
    VolumeDesc::numPBlocks = VolumeDesc::totalSize / VolumeDesc::pBlockSize;

    // parent-block XY plane size
    VolumeDesc::bPlaneSize = {
    	VolumeDesc::totalSize.x,
    	VolumeDesc::totalSize.y,
    	VolumeDesc::pBlockSize.z };

    // volume of a parent block
    VolumeDesc::bPlaneNumVoxels = getVolume(VolumeDesc::bPlaneSize);
}

// reserve memory to store Volume on host and device
__host__ void reserveVolumeMemory() 
{
    // Reserve pinned memory for each 
    blockPlane.reserve(VolumeDesc::numPBlocks.z);

    for (ushort i = 0; i < VolumeDesc::numPBlocks.z; i++)
    {
        uint* data;
        hipHostMalloc((void**)&data, VolumeDesc::bPlaneNumVoxels * sizeof(uint));

        // calculate offset to origin of block plane
        ushort3 pBlockIndex = { 0, 0, i };
        ushort3 offset = pBlockIndex * VolumeDesc::bPlaneSize;

        blockPlane.push_back({ data, offset });
    }
}

// Initialise the compression vars and TagReader
__host__ void setup()
{
    readFileDescription();
    reserveVolumeMemory();
}

// Cleanup any memory allocated on the heap
__host__ void cleanup()
{
    // host copy of all block planes in the volume
    for (const BlockPlane& bPlane : blockPlane)
    {
        hipHostFree(bPlane.volume);
    }
}

// read voxels from file
__host__ void readVolume()
{
    // for each block plane
    for (ushort i = 0; i < VolumeDesc::numPBlocks.z; i++)
    {
        // read number of voxels that can fit in block plane
        for (ulong j = 0; j < VolumeDesc::bPlaneNumVoxels; j++)
        {

            uchar tagID = globalTT.getID(TagReader::getNextTagName());

            // store in block plane
            blockPlane[i].volume[j] = tagID;
        }
    }
}

int main()
{
    // Check there is an available CUDA enabled device
    hipSetDevice(0);
    cudaCheckErrors("set Device failed");

    // initialise program
    setup();
    Compressor::setupDevice(VolumeDesc::totalSize, VolumeDesc::pBlockSize);

    // read voxels into block Planes
    readVolume();

    // compress block planes
    for (int i=0; i<blockPlane.size(); i++)
    {
        Compressor::compressPrint(blockPlane[i]);
    }

    // cleanup any allocated memory
    cleanup();
    Compressor::cleanupDevice();

#ifdef USE_TIMER
    cout << "Compression Time: " << timerTotal << endl;
#endif

    return 0;
}