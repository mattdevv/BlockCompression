#include "hip/hip_runtime.h"
#include "Compressor.cuh"

#define BYTES_IN_BLOCK 16
struct __align__(BYTES_IN_BLOCK) Block
{
    ushort3 size;   // 6 bytes
    ulong index;    // 8 bytes
    uchar ID;       // 1 byte
    bool active;    // 1 byte
};

namespace Compressor
{
	const uint KERNEL_SIZE = 64;            // number of kernels in a thread group
	const uint NULL_INDEX = 0xFFFFFFFF;     // uint value representing index of a NULL block
    
    // device global constants
    __constant__ uint3 d_maxThreads;        // how many threads should be run for a Kernel acting on dimension XYZ
	__constant__ ushort3 d_volDimensions;   // dimension sizes of volume loaded on device (currently a block plane)
    __constant__ ushort3 d_pBlockSize;      // size of a parent block
	__constant__ uint3 d_translations;      // precalculated values to move through 3D volume using 1D offsets

    // device pointers to device variables
    uint* dev_numBlocks;				    // counter of found blocks
    uint* dev_nextBlockIndex;			    // counter for current block index
    Block* dev_blockList;				    // pointer to list of blocks
    uint* dev_volume;					    // pointer to loaded volume

    // host variables
    Block* blockList;                       // host-side array of blocks used as location to copy from device into
    uint3 threadsPerDimension;              // how many threads should be run when compressing dimension X, Y, or Z

	uint numBlocks = 0;                     // maximum number of Blocks (a struct) in the current loaded volume
											// type is uint, so maximum number of blocks must not exceed 2^32
											// can happen if volume size is >1600x1600x1600 and no compression possible
											// storing 2^32 blocks would take 65GB though so the program still wouldnt work
											// recommend keeping size less than 1000x1000x1000 and a volume of 1 billion
											// at this size the worst case memory usage is ~20GB

    ushort3 volumeSize;                     // size of entire volume
    ushort3 pBlockSize;                     // size of parent block cells inside volume

    ushort3 deviceVolumeSize;               // exact XYZ size of volume to load to device, does not change
											// not size of whole volume, just amount to load to device at once
	ull deviceVolumeNumVoxels;              // count of how many voxels fit in deviceVolumeSize
											// used to know how many bytes to copy


    // kernel to count maximum number of blocks that will be outputted
    // used to allocate memory for blocks and assign an index to new blocks
    __global__ void countMinimumBlocks(uint* volume, uint* numBlocks)
    {
        // unique thread index from thread group number and interior threadID
        ulong threadID = blockIdx.x * blockDim.x + threadIdx.x;

        // kill thread if index is more than needed amount
        if (threadID >= d_maxThreads.x)
            return;

        // convert threadID to 1D position in volume
        // only places threads along the first YZ slice of parent blocks
        ulong planeVolume = d_volDimensions.y * d_volDimensions.z;
        ulong planeIndex = threadID / planeVolume;
        threadID = threadID % planeVolume;

        // starting 1D position in parent block (YZ slice at local X=0)
        ulong threadPos1D = threadID * d_volDimensions.x + planeIndex * d_pBlockSize.x;

        // find where to stop
        ulong endPos1D = threadPos1D + d_pBlockSize.x - 1;

        // start foundBlocks at one as the final voxel is skipped due to out-of-range
        uint foundBlocks = 1;

        while (threadPos1D < endPos1D)
        {
            // count how many times two sequential tags are not equal
            if (volume[threadPos1D] != volume[threadPos1D + 1])
            {
                foundBlocks++;
            }
            threadPos1D++;
        }

        // thread-safe addition to global counter
        atomicAdd(numBlocks, foundBlocks);
    }

    // kernel to perform x compression and create the first list of blocks
    // loaded volume is also converted from representing tags, to the index in the block-list occupying that space
    // a null tag is used for all voxels besides the block's origin so the block can only be accessed from reading one voxel
    __global__ void xRowCompression(uint* volume, Block* blockList, uint* nextBlockIndex)
    {
        // unique thread index from thread group number and interior threadID
        ulong threadID = blockIdx.x * blockDim.x + threadIdx.x;

        // kill thread if index is more than needed amount
        if (threadID >= d_maxThreads.x)
            return;

        // calculate 1D position
        ulong planeVolume = d_volDimensions.y * d_volDimensions.z;
        ulong planeIndex = threadID / planeVolume;
        threadID = threadID % planeVolume;

        // 1D position in volume (YZ slice at X=0 for some parent block)
        ulong threadPos1D = threadID * d_volDimensions.x + planeIndex * d_pBlockSize.x;

        // find where to stop
        ulong endPos1D = threadPos1D + d_pBlockSize.x;

        // start the first block line
        ushort length = 1;
        uchar prevTag = volume[threadPos1D];
        uint blockOrigin1D = threadPos1D;

        // move over first voxel which was already read
        threadPos1D++;

        // thread moves along x axis for length of parent block X
        while (threadPos1D < endPos1D)
        {
            uchar thisTag = volume[threadPos1D];

            // grow line if same
            if (thisTag == prevTag)
            {
                length++;
            }
            // break line if different
            else
            {
                // request a blockList index
                uint blockIndex = atomicAdd(nextBlockIndex, 1);

                // backtrack and set voxels to null
                // origin is set to be the block's index
                volume[blockOrigin1D] = blockIndex;
                for (ushort j = 1; j < length; j++)
                {
                    volume[blockOrigin1D + j] = NULL_INDEX;
                }

                // store the block in the list
                blockList[blockIndex] = {
                    { length, 1, 1 },
                    blockOrigin1D,
                    prevTag,
                    true };

                // reset line
                prevTag = thisTag;
                length = 1;
                blockOrigin1D = threadPos1D;
            }

            // move along X once
            threadPos1D++;
        }

        // manually create the final block
        // request a blockList index
        uint blockIndex = atomicAdd(nextBlockIndex, 1);

        // backtrack and set voxels to null
        // origin is set to be the block's index
        volume[blockOrigin1D] = blockIndex;
        for (ushort j = 1; j < length; j++)
        {
            volume[blockOrigin1D + j] = NULL_INDEX;
        }

        __syncthreads();

        // store the block in the list
        blockList[blockIndex] = {
            { length, 1, 1 },
            blockOrigin1D,
            prevTag,
            true };
    }

    // kernel to perform y compression
    // blocks eliminated by compression are removed from volume but still in block-list marked as inactive 
    __global__ void yRowCompression(uint* volume, Block* blockList)
    {
        // unique thread index from thread group number and interior threadID
        ulong threadID = blockIdx.x * blockDim.x + threadIdx.x;
        // kill thread if higher than pre-calculated allowed amount
        if (threadID >= d_maxThreads.y)
            return;

        // variables to help find thread position
        const ulong planeVolume = d_volDimensions.x * d_volDimensions.z;
        const ulong planeIndex = threadID / planeVolume;
        const uint pBlockYTranslation = d_pBlockSize.y * d_translations.y;
        threadID = threadID % planeVolume;

        // find thread's individual offsets per dimension
        const ulong offset1DX = threadID % d_volDimensions.x;
        const ulong offset1DY = planeIndex * pBlockYTranslation;
        const ulong offset1DZ = (threadID / d_volDimensions.x) * d_translations.z;

        // 1D position in volume (XZ slice at Y=0 for some parent block)
        ulong threadPos1D = offset1DX + offset1DY + offset1DZ;

        ulong endPos1D = threadPos1D + pBlockYTranslation;

        Block* p_blockBelow;
        Block blockBelow;
        bool canMergeBelow = false;

        // thread moves along y axis for length of parent block Y
        while (threadPos1D < endPos1D)
        {
            // get reference to block at thread position
            uint blockIndex = volume[threadPos1D];

            // if null, thread is not inside the origin of a block
            // must be inside origin for thread to be allowed to use that block
            if (blockIndex == NULL_INDEX)
            {
                canMergeBelow = false;
                threadPos1D += d_translations.y;
                continue;
            }

            Block* p_block = &blockList[blockIndex];
            Block block = blockList[blockIndex];

            if (canMergeBelow 
                && block.size.x == blockBelow.size.x 
                && block.ID == blockBelow.ID)
            {
                // update blocks
                p_block->active = false;
                p_blockBelow->size.y++;

                // disable top block in index volume so Z pass doesn't use it
                volume[threadPos1D] = NULL_INDEX;
            }
            else
            {
                p_blockBelow = p_block;
                blockBelow = block;
                canMergeBelow = true;
            }

            threadPos1D += d_translations.y;
        }
        
    }

    // kernel to perform z compression
    // blocks eliminated by compression are kept in the volume and still in block-list marked as inactive 
    __global__ void zRowCompression(uint* volume, Block* blockList)
    {
        // unique thread index from thread group number and interior threadID
        ulong threadID = blockIdx.x * blockDim.x + threadIdx.x;
        // kill thread if higher than pre-calculated allowed amount
    	if (threadID >= d_maxThreads.z)
            return;

        // 1D position in volume (XY slice at Z=0, always bottom parent block)
        ulong threadPos1D = threadID;

        // calculate maximum offset from starting position
        ulong endPos1D = threadPos1D + d_volDimensions.z * d_translations.z;

        // store a previous block
        Block* p_blockBelow;
        Block blockBelow;
        bool canMergeBelow = false; // check if stored block is valid merge

        // thread moves along z axis for length of parent block Z
        while (threadPos1D < endPos1D)
        {
            // get reference to block at thread position
            uint blockIndex = volume[threadPos1D];

            // if null, thread is not inside the origin of a block
            // must be inside origin for thread to be allowed to use that block
            if (blockIndex == NULL_INDEX)
            {
                canMergeBelow = false;
                threadPos1D += d_translations.z;
                continue;
            }

            // index not null, grab current block pointer + copy
            Block* p_block = &blockList[blockIndex];
            Block block = blockList[blockIndex];

            // if last block was a valid merge
            // and both blocks have same XY dimensions
            // and both blocks have same tagIDs
            if (canMergeBelow 
                && block.size.x == blockBelow.size.x 
                && block.size.y == blockBelow.size.y 
                && block.ID == blockBelow.ID)
            {
                // perfect match => can merge top block into bottom block
                p_block->active = false;
                p_blockBelow->size.z++;
                // we dont need to update volume as no other threads will execute
                //volume[threadPos1D] = NULL_INDEX;
            }
            else
            {
                // did not match but top block might match next block so store it
                p_blockBelow = p_block;
                blockBelow = block;
                canMergeBelow = true;
            }

            // move along Z once 
            threadPos1D += d_translations.z;
        }
    }

    // set constant variables on device
    __host__ void setupDeviceConstants()
    {
    	// Calculate how many 1D voxels to move over for a single translation in each 3D dimension
        uint3 translations = {
            (uint)1, // one voxel
            (uint)deviceVolumeSize.x, // one x row
            (uint)(deviceVolumeSize.x * deviceVolumeSize.y) // one xy plane
        }; 

        // Calculate how many voxels exist in a single slice per dimension
        uint3 sliceVolumes = {
            deviceVolumeSize.y * deviceVolumeSize.z,
            deviceVolumeSize.x * deviceVolumeSize.z,
            deviceVolumeSize.x * deviceVolumeSize.y
        };

        // how many parent-blocks are there in a block plane
        uint3 blocksInVolume = {
        	(uint)(volumeSize.x / pBlockSize.x),
            (uint)(volumeSize.y / pBlockSize.y),
        	1
        };

        // start a thread in every voxel of 1 slice in every parent-block
        // slice is along dimension being compressed, e.g when X slice is YZ plane
        threadsPerDimension = blocksInVolume * sliceVolumes;

        // copy constants to device
        hipMemcpyToSymbol(HIP_SYMBOL(d_maxThreads), &threadsPerDimension, sizeof(uint3));
        cudaCheckErrors("set d_maxPlaneIndex failed");
        hipMemcpyToSymbol(HIP_SYMBOL(d_volDimensions), &deviceVolumeSize, sizeof(ushort3));
        cudaCheckErrors("set d_volDimensions failed");
        hipMemcpyToSymbol(HIP_SYMBOL(d_pBlockSize), &pBlockSize, sizeof(ushort3));
        cudaCheckErrors("set d_pBlockDimensions failed");
        hipMemcpyToSymbol(HIP_SYMBOL(d_translations), &translations, sizeof(uint3));
        cudaCheckErrors("set d_translations failed");
    }

    // allocate memory for frequently used variables
    __host__ void reserveDeviceMemory()
    {
        // Reserve memory for the volume on GPU device
        hipMalloc((void**)&dev_volume, getVolume(deviceVolumeSize) * sizeof(uint));
        cudaCheckErrors("malloc device volume failed");

        // counter for the maximum number of blocks
        hipMalloc((void**)&dev_numBlocks, 1 * sizeof(uint));
        cudaCheckErrors("malloc device numBlocks failed");

        // counter for the number of blocks that have been created
        hipMalloc((void**)&dev_nextBlockIndex, 1 * sizeof(uint));
        cudaCheckErrors("malloc device nextBlockIndex failed");
    }

    // do all setup to start using device
    __host__ void setupDevice(ushort3& descriptionVolumeSize, ushort3& descriptionParentBlockSize)
    {
        volumeSize = descriptionVolumeSize;
        pBlockSize = descriptionParentBlockSize;

        // device will read an XY plane 1 parent block thick along Z
        deviceVolumeSize = { volumeSize.x, volumeSize.y, pBlockSize.z };
        deviceVolumeNumVoxels = getVolume(deviceVolumeSize);

        setupDeviceConstants();
        reserveDeviceMemory();
    }

    // move a volume of voxels onto device
    __host__ void copyVolumeToDevice(uint* volume)
    {
        // Copy volume of voxels from host to device.
        hipMemcpy(dev_volume, volume, deviceVolumeNumVoxels * sizeof(uint), hipMemcpyHostToDevice);
        cudaCheckErrors("memcpy volume to device failed");
    }

    // reset counters for a new parent block
    __host__ void resetBlockCounters()
    {
        const uint zero = 0;
        hipMemcpy(dev_numBlocks, &zero, sizeof(uint), hipMemcpyHostToDevice);
        cudaCheckErrors("device set numBlocks=0 failed");
        hipMemcpy(dev_nextBlockIndex, &zero, sizeof(uint), hipMemcpyHostToDevice);
        cudaCheckErrors("device set nextBlockIndex=0 failed");
    }

    // creates array to hold the outputted blocks
	// size of array is calculated by a kernel that sweeps the entire volume
    __host__ void createBlockList()
    {
        // set values in device counters to 0
        resetBlockCounters();

        static const uint numThreadBlocks = divideCeil(threadsPerDimension.x, KERNEL_SIZE);

        // count number of blocks needed for run-length encoding
        countMinimumBlocks <<< numThreadBlocks, KERNEL_SIZE >>> (dev_volume, dev_numBlocks);
        cudaCheckErrors("countMinimumBlocks kernel failed");

        // copy the counter to host
        hipMemcpy(&numBlocks, dev_numBlocks, sizeof(uint), hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy numBlocks failed");
    	//cout << "Num Blocks: " << numBlocks << endl;

        // Reserve memory for the blockList on host and device
        blockList = new Block[numBlocks];
        hipMalloc((void**)&dev_blockList, numBlocks * BYTES_IN_BLOCK);
        cudaCheckErrors("malloc device blockList failed");
    }

    // free device and host memory containing block-list
    __host__ void deleteBlockList()
    {
        delete[] blockList;
        hipFree(dev_blockList);
    }

    // after all variables have been setup compress the voxels
    __host__ void compressVolume()
    {
        // compress along X
        static const uint numBlocksX = divideCeil(threadsPerDimension.x, KERNEL_SIZE);
        xRowCompression <<< numBlocksX, KERNEL_SIZE >>> (dev_volume, dev_blockList, dev_nextBlockIndex);
        cudaCheckErrors("xrow compression kernel failed");

        // compress along Y
        static const uint numBlocksY = divideCeil(threadsPerDimension.y, KERNEL_SIZE);
        yRowCompression <<< numBlocksY, KERNEL_SIZE >>> (dev_volume, dev_blockList);
        cudaCheckErrors("yrow compression kernel failed");

        // compress along Z
        static const uint numBlocksZ = divideCeil(threadsPerDimension.z, KERNEL_SIZE);
        zRowCompression <<< numBlocksZ, KERNEL_SIZE >>> (dev_volume, dev_blockList);
        cudaCheckErrors("zrow compression kernel failed");
    }
    
    // convert 1D position to 3D
    __host__ ushort3 convert1DIndexTo3D(const ulong index)
    {
        static const uint xyPlaneVolume = (deviceVolumeSize.x * deviceVolumeSize.y);

        const ushort xDirection = index % deviceVolumeSize.x;
        const ushort yDirection = (index / deviceVolumeSize.x) % deviceVolumeSize.y;
        const ushort zDirection = index / xyPlaneVolume;

        return { xDirection, yDirection, zDirection };
    }

    // copy all blocks to host for outputting
    // some invalid blocks will be mixed in marked as inactive
    __host__ void copyBlocksToHost()
    {
        hipMemcpy(blockList, dev_blockList, numBlocks * BYTES_IN_BLOCK, hipMemcpyDeviceToHost);
        cudaCheckErrors("memcpy blockList to host failed");
    }

    // write to console the active blocks
    __host__ void printBlocks(const ushort3& offset)
    {
        // print the valid blocks
        for (int i = 0; i < numBlocks; i++)
        {
            const Block& block = blockList[i];

            if (block.active)
            {
                // block only stores 1D position
                // convert back to 3D and add blockPlane offset
                const ushort3 origin = offset + convert1DIndexTo3D(block.index);

                // convert tagIDs back to strings using a tagTable
                std::cout << coordToString(origin) + coordToString(block.size)
                    + "\'" + globalTT.getTag(block.ID) + "\'\n";
            }
        }
    }

    // deallocate reused memory at program end
    __host__ void cleanupDevice()
    {
        // free device variables that are kept between calls
        hipFree(dev_volume);
        hipFree(dev_numBlocks);
        hipFree(dev_nextBlockIndex);
    }

    // after calling setupDevice only need an array of voxels + an offet
    // includes all steps to perform compression and printing + memory management
    __host__ void compressPrint(BlockPlane& pBlock)
    {
        // copy voxels to device
        copyVolumeToDevice(pBlock.volume);

#ifdef USE_TIMER
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif

        /* a pre-pass counts the maximum blocks that compression will 
        result in and allocates block list to hold output blocks */
        createBlockList();

        // do 3 passes of compression
        compressVolume();

#ifdef USE_TIMER
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        timerTotal += milliseconds;
#else

    	// copy the list of all blocks
        copyBlocksToHost();

        // copy blocks back to host and print
        printBlocks(pBlock.offset);

#endif

        // cleanup results of this volume
        deleteBlockList();
    }
}